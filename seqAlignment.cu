#include "hip/hip_runtime.h"
//#include "stdafx.h"
// Sequence Alignment -CUDA
// Alex Ringeri

//C Libraries
#include <stdio.h>
#include <string.h>
#include <stdlib.h>
#include <math.h>

//CUDA Libraries
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>



#define numThreads 128

void testReduce();

void CalculateCost(int *d_matrix, int *d_trace, char *d_s1, char *d_s2, int s1, int s2);

void CopyToMatrix(int *dst, int *src, int cols, int rows);

void PrintMatrix(int *arr, char *s1, char *s2, int xDim, int yDim);

//Kernel initializes all elements of matrix to 'value'
__global__ void init_matrix(int *matrix, int value, int maxElements){
	if (blockDim.x * blockIdx.x + threadIdx.x < maxElements)
		matrix[ blockDim.x * blockIdx.x + threadIdx.x] = value;
}

//Finds max value of array and places its index into '*iOut'
//Credit to udacity - Lesson 3 - reduction
__global__ void maxReduce(int *table, int *maxOut, int *iOut){
	int id = threadIdx.x;
	int g_id = blockIdx.x*blockDim.x*2 +threadIdx.x;
	extern __shared__ int s_table[];
	
	//copy from global to shared memory using all threads in block 
	//And do first reduce
	if (table[g_id] > table[g_id + id]){
		s_table[g_id] = table[g_id + id];
		//store indices each larger element in 2nd unused part of array
		s_table[id + blockDim.x] = g_id + id;
	}
	else{
		s_table[id + blockDim.x] = id;
	}
	syncthreads(); 
	
	//Do first reduction and store indices in 2nd half of array
	//May only need the for loop below and output
	unsigned int i = blockDim.x/2;
	if (id < i){
		if (s_table[id+i] > s_table[id]){
			s_table[id] = s_table[id + i];
			s_table[id+i] = id + i;
		}
		else
			s_table[id+i] = id;
		syncthreads();
		/*if (id ==0){
			for (int j=0; j < blockDim.x; j++)
				printf("%d ", s_table[j]);
			printf("\n");
		}
		syncthreads();*/
	}
	
	int temp = i;
	for(i >>= 1; i > 0; i>>=1){
		if (id < i){
			if (s_table[id+i] > s_table[id]){
				s_table[id] = s_table[id + i];
				s_table[id+i] = s_table[id+i+temp];
			}
			else
				s_table[id+i] = s_table[id + temp];
		}
		temp = i;
		syncthreads();
		/*if (id ==0){
			for (int j=0; j < blockDim.x; j++)
				printf("%d ", s_table[j]);
			printf("\n");
		}
		syncthreads();*/
	}
	
	//place resulting index into out
	if (id == 0){
		maxOut[blockIdx.x] = s_table[id];
		iOut[blockIdx.x] = s_table[id+1];
		//printf("BlockId: %d\tMax: %d\tIndex: %d\n", blockIdx.x, s_table[id], s_table[id+1]);
	}
}

void testReduce(int size){

	//scanf("%d", &size);
	int *arr = (int*)malloc(sizeof(int)*size);
	srand(time(NULL));
	for (int i=0; i <size; i++)
		arr[i] = rand()%500;
	
	int *d_i, *d_arr, *d_max;
	 
	hipMalloc((void**)&d_arr, sizeof(int)*size);
	hipMemcpy(d_arr, arr, sizeof(int)*size, hipMemcpyHostToDevice);
	
	int threadsPerBlock = 1024;
	int blocks = (size + threadsPerBlock - 1)/threadsPerBlock;
	hipMalloc((void**)&d_i, sizeof(int)*blocks); 
	hipMalloc((void**)&d_max, sizeof(int)*blocks);	
	

	maxReduce<<< blocks, threadsPerBlock, sizeof(int)*threadsPerBlock>>>(d_arr, d_max, d_i);
	hipDeviceSynchronize();
	
	int *intermediateIndex = (int*)malloc(sizeof(int)*blocks);
	hipMemcpy(intermediateIndex, d_i, sizeof(int)*blocks, hipMemcpyDeviceToHost);
	
	int *j = (int*)malloc(sizeof(int));
	hipMemcpy(j, d_max, sizeof(int)*blocks, hipMemcpyDeviceToHost);

	if (size >1024){
		threadsPerBlock = blocks;
		blocks = 1;
		int *d_maxVal; int *d_interI;
		hipMalloc((void**)&d_maxVal, sizeof(int)); 
		hipMalloc((void**)&d_interI, sizeof(int));
		maxReduce<<< blocks, threadsPerBlock, sizeof(int)*threadsPerBlock>>>(d_max, d_maxVal, d_interI);
		hipDeviceSynchronize();
		
		int *i = (int*)malloc(sizeof(int));

		hipMemcpy(i, d_interI, sizeof(int), hipMemcpyDeviceToHost);
		hipMemcpy(j, d_maxVal, sizeof(int), hipMemcpyDeviceToHost);

		printf("Kernel: max %d index %d\n", *j, intermediateIndex[*i]);
		int max = 0; int in = 0;
		for (int k=0; k <size; k++){
			if (arr[k] >= max){
				max = arr[k];
				in = k;
			}
		}
		printf("Host: max %d index %d\n", max, in);
		free(i);
		free(j);
		hipFree(d_maxVal); 
		hipFree(d_interI);
	}
	else{
		printf("Kernel: max %d index %d\n", *j, *intermediateIndex);
		int max = 0; int in = 0;
		for (int k=0; k <size; k++){
			if (arr[k] >= max){
				max = arr[k];
				in = k;
			}
		}
		printf("Host: max %d index %d\n", max, in);
		free(j);
	}
	
	free(arr); free(intermediateIndex);
	hipFree(d_arr); hipFree(d_i); hipFree(d_max); 
}

__global__ void ComputeDiagonal(int i, int prevI, int lastI, int space, int *arr, int *trace, char *s1, char *s2, int s1off, int s2off){
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	if (id < space){
		int left = arr[prevI + id];
		int up = arr[prevI + id + 1];
		int upLeft = arr[lastI + id];
		
		if(s1[s1off + id] == s2[s2off - id] )
			upLeft += 2;
		else
			upLeft -= 1;

		int cost, dir;
		if (up > left){
			cost = up - 1;
			dir = 1;
		}
		else{
			cost = left - 1;
			dir = -1;
		}
		if (upLeft > cost){
			cost = upLeft;
			dir = 0;
		}
		arr[i + id] = max(cost, 0);
		trace[i + id] = i+id;// dir;	
	}
}

__global__ void CalculateCostOneKernel(int *d_matrix, int *d_trace, char *d_s1, char *d_s2, int s1, int s2){
	int i = 3;
	int prev = 1;
	int last = 0;

    for (int slice = 2; slice < s2 + s1 - 1; slice++) {
        int z1 = 0;
		int z2 = 0;
		int numElements = 0;
		int off = 1;
		if (slice > s1-1){
			z1 = slice - s1 + 1;
			numElements++;
		}
		if (slice > s2-1){
			z2 = slice - s2 + 1;
			numElements++;
			off = 0;
		}
        int size = slice - z1 - z2 +1;
		numElements += size -2;
		if (z2>1) last++;
		
		for (int s = 0; s < (numElements + blockDim.x -1)/blockDim.x; s++){
			int id = blockDim.x * s + threadIdx.x;
			if (id < numElements){
				int upLeft = d_matrix[last + id];
				int left = d_matrix[prev + id];
				int up = d_matrix[prev + id + 1];
				
				if(d_s1[max(z2-1, 0) + id] == d_s2[min(slice-2, s2-2) - id] )
					upLeft += 2;
				else
					upLeft -= 1;

				int cost, dir;
				if (up > left){
					cost = up - 1;
					dir = 1;
				}
				else{
					cost = left - 1;
					dir = -1;
				}
				if (upLeft > cost){
					cost = upLeft;
					dir = 0;
				}
				d_matrix[i + off + id] = max(cost, 0);
				d_trace[i + off + id] = dir;
			}
		}
		last = prev;
		prev = i;
		i += size;
		syncthreads();
    }
}


// Main routine: Executes on the host
int main(int argc, char *argv[]){
	char AGCT[] = "AGCT";
	int lenS1, lenS2;
	if (argc > 2){
		int args[] = { atoi(argv[1]), atoi(argv[2]) };
		if (args[0] > args[1]){
			lenS1 = args[0];
			lenS2 = args[1];
		}else {
			lenS1 = args[1];
			lenS2 = args[0];
		}
	}else {
		printf("Invalid Command Line Arguments --- Exiting Program");
		exit(0);
	}

	//Allocate strings on host
	char * string1 = (char*)malloc(sizeof(char)*lenS1);
	char * string2 = (char*)malloc(sizeof(char)*lenS2);

	//Initialize strings with random numbers
	srand(time(NULL));
	for(int i=0; i<lenS1 ;i++)
		string1[i] = AGCT[rand()%4];
	for(int i=0; i<lenS2 ;i++)
		string2[i] = AGCT[rand()%4];
	
	//Allocate strings on device
	hipError_t error = hipSuccess;
	char *d_string1, *d_string2;
	
	error = hipMalloc((void**)&d_string1, sizeof(char)*lenS1);
	
	if (error != hipSuccess) {
		printf("Error allocating s1 on device\n");
		exit(0);
	}
	
	error = hipMalloc((void**)&d_string2, sizeof(char)*lenS2);
	
	if (error != hipSuccess) {
		printf("Error allocating s2 on device\n");
		exit(0);
	}
	
	//Initialize sequence strings on device
	error = hipMemcpy(d_string1, string1, sizeof(char)*lenS1, hipMemcpyHostToDevice);
	
	if (error != hipSuccess) {
		printf("Error copying s1 to device\n");
		exit(0);
	}
	
	error = hipMemcpy(d_string2, string2, sizeof(char)*lenS2, hipMemcpyHostToDevice);
	
	if (error != hipSuccess) {
		printf("Error copying s2 to device\n");
		exit(0);
	}

	//Allocate score table on Device
	int entries = (lenS1+1)*(lenS2+1);
	int* d_matrix;
	error = hipMalloc((void**)&d_matrix, sizeof(int)*entries);
	
	if (error != hipSuccess) {
		printf("Error allocating d_matrix on device\n");
		exit(0);
	}

	int threadsPerBlock = 256;
	int blocksPerGrid = (entries + threadsPerBlock -1)/threadsPerBlock;
	
	//Initialize score table with 0
	init_matrix<<< blocksPerGrid, threadsPerBlock >>>(d_matrix, 0, entries);
	
	
	//Allocate trace table on Device
	int* d_trace;
	error = hipMalloc((void**)&d_trace, sizeof(int)*entries);
	
	if (error != hipSuccess) {
		printf("Error allocating d_trace on device\n");
		exit(0);
	}

	//Initialize trace table with -2
	init_matrix<<< blocksPerGrid, threadsPerBlock >>>(d_trace, -2, entries);

	/* Do calculation on device:
	 *
	 */

	CalculateCost(d_matrix, d_trace, d_string1, d_string2, lenS1+1, lenS2+1);
	hipDeviceSynchronize();
	
	error = hipGetLastError();
	
	if (error != hipSuccess) {
		printf("Error with kernel or d_matrix/d_trace allocation: %s\n", hipGetErrorString(error));
		exit(0);
	}
	
	//testReduce(atoi(argv[3]));
	/*int *posMax;
	error = hipMalloc((void**)&posMax, 1*sizeof(int));
	maxReduce<<< 1, 1024, 2048*sizeof(int) >>>(d_matrix, posMax);
	
	hipDeviceSynchronize();
	int *pos = (int*)malloc(1*sizeof(int));
	hipMemcpy(pos, posMax, 1*sizeof(int), hipMemcpyDeviceToHost);*/

	//Allocate and copy score table to host
	int *k1Result =(int*)malloc(sizeof(int)*entries);
	hipMemcpy(k1Result, d_matrix, sizeof(int)*entries, hipMemcpyDeviceToHost);
	
	//Allocate final matrix: Used for output (easier printing)
	int *matrix2d = (int*)malloc(sizeof(int)*entries);
	CopyToMatrix(matrix2d, k1Result, lenS1+1, lenS2+1);
	
	int *trace =(int*)malloc(sizeof(int)*entries);
	hipMemcpy(trace, d_trace, sizeof(int)*entries, hipMemcpyDeviceToHost);
	
	if (argc > 3 && !strcmp("-v",argv[3])){ 
		printf("Kernel 1: Less work per kernel:\n");
		PrintMatrix(matrix2d, string1, string2, lenS1+1, lenS2+1);
	}
	
	threadsPerBlock = 256;
	blocksPerGrid = (entries + threadsPerBlock -1)/threadsPerBlock;
	
	//Initialize score table with 0
	init_matrix<<< blocksPerGrid, threadsPerBlock >>>(d_matrix, 0, entries);
	
	hipDeviceSynchronize();
	
	int *k2Result = (int*)malloc(sizeof(int)*entries);
	CalculateCostOneKernel<<< 1, 256 >>>(d_matrix, d_trace, d_string1, d_string2, lenS1+1, lenS2+1);
	hipDeviceSynchronize();
	hipMemcpy(k2Result, d_matrix, sizeof(int)*entries, hipMemcpyDeviceToHost);
	CopyToMatrix(matrix2d, k2Result, lenS1+1, lenS2+1);


	
	if (argc > 3 && !strcmp("-v",argv[3])){ 
		printf("\n\n2nd Kernel: More work per kernel\n");
		PrintMatrix(matrix2d, string1, string2, lenS1+1, lenS2+1);
	}
	
	int res = 1;
	for (int i =0; i < entries; i++){
		if (k1Result[i] != k2Result[i]){
			res = 0;
			printf("i: %d\t%d\t%d\n", i,k1Result[i], k2Result[i]);
		}
	}
	if (res)
		printf("Kernel 1 matches Kernel 2\n");
	else
		printf("Kernel 1 does not match Kernel 2\n");
	
	

	//Allocate and copy trace table to host
	CopyToMatrix(matrix2d, trace, lenS1+1, lenS2+1);
	//PrintMatrix(matrix2d, string1, string2, lenS1+1, lenS2+1);


	//This Section causes an error:  "object was probably modified after being freed"
	/**Find largest value in matrix and then walk back until a '0' <- matrix[ix+j] value is found.
	 Find local alignment
	int max = 0;
	int maxPos = 0;
	for(i=0; i < (lenS1+1)*(lenS2+1);i++){
		if (matrix[i] > max){
			max = matrix[i];
			maxPos = i;
		}
	}

	int length = 0;
	i=maxPos/(lenS1+1) ;
	int j = maxPos%(lenS2+1);
	char *finalS1 = (char*)malloc(sizeof(i+j));
	char *finalS2 = (char*)malloc(sizeof(i+j));

	printf("max: %d\tmaxPos: %d\n", max, maxPos);
	while (matrix[i*(lenS2+1)+j] > 0){
		int dir = trace[i*(lenS2+1)+j];
		if (dir == -1){
			i--;
			finalS1[length] = string1[i];
			finalS2[length++] = '-';
		}else if (dir == 0){
			i--;j--;
			finalS1[length] = string1[i];
			finalS2[length++] = string2[j];
		}else{
			j--;
			finalS2[length] = string2[i];
			finalS1[length++] = '-';
		}
	}
	//printf("String1: %s\nString2: %s\n", finalS1, finalS2);
	for (i=length-1; i >= 0; i--){
		printf("%c",finalS1[i]);
	}
	printf("\n");
	for (i=length-1; i >= 0; i--){
		printf("%c",finalS2[i]);
	}
	free(finalS1);
	free(finalS2);
	*/

	hipDeviceSynchronize();
	//Free device memory
	hipFree(d_string1);
	hipFree(d_string2);
	hipFree(d_matrix);
	hipFree(d_trace);
	//hipFree(posMax);
	
	//Free host memory
	free(string1);
	free(string2);
	free(matrix2d);
	free(k1Result);
	free(k2Result);
	free(trace);	

	hipDeviceReset();
}


/**	
*
*/
void CalculateCost(int *d_matrix, int *d_trace, char *d_s1, char *d_s2, int s1, int s2){
	int i = 3;
	int prev = 1;
	int last = 0;

	hipStream_t stream1;
	hipStream_t stream2;
	hipStreamCreate(&stream1);
	hipStreamCreate(&stream2);

    for (int slice = 2; slice < s2 + s1 - 1; slice++) {
        int z1 = slice < s1 ? 0 : slice - s1 + 1;
        int z2 = slice < s2 ? 0 : slice - s2 + 1;
        
		int size = slice - z1 - z2 +1;
		int numElements = size -2;
		
		if (z2>1) last++;
		if (z1 > 0) numElements++;
		
		int off =1;
		if (z2 > 0) { numElements++; off = 0; };
		int blocksPerGrid = (numElements + numThreads - 1)/numThreads;
        
		ComputeDiagonal<<<blocksPerGrid, numThreads, 0, stream1>>>(i + off, prev, last, numElements, d_matrix, d_trace, d_s1, d_s2, max(z2-1, 0), min(slice-2, s2-2));

		ComputeDiagonal<<< blocksPerGrid, numThreads, 0, stream2>>>
				(i + off, prev, last, numElements, d_matrix, d_trace, d_s1, d_s2, max(z2-1, 0), min(slice-2, s2-2));
		last = prev;
		prev = i;
		i += size;
    }
    hipStreamDestroy(stream1);
    hipStreamDestroy(stream2);
}

void PrintMatrix(int *arr, char *s1, char *s2, int xDim, int yDim){
	printf("\t");
	for(int i = 0; i < xDim - 1; i++){
		printf("\t%c", s1[i]);
	}
	printf("\n------------------------------------------------------------------------------------------------------\n\t|");
	for(int i =0; i < yDim; i++){	
		for(int j = 0; j < xDim; j++)
			printf("%d\t",arr[i*xDim + j]);
		printf("\n%c\t|", s2[i]);
	}printf("\n");
}

void CopyToMatrix(int *dst, int *src, int cols, int rows){
	/**Credit Mark Byers at Stack overflow: http://stackoverflow.com/a/2112951 */
	int i = 0;
	for (int slice = 0; slice < cols + rows - 1; ++slice) {
        int z1 = slice < cols ? 0 : slice - cols + 1;
        int z2 = slice < rows ? 0 : slice - rows + 1;
        for (int j = slice - z2; j >= z1; --j) {
            dst[cols*j + slice - j] = src[i++];
        }
    }
}


